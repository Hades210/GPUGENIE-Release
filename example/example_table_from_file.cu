#include "hip/hip_runtime.h"
/*
 * example_tweets.cu
 *
 *  Created on: Oct 16, 2015
 *      Author: zhoujingbo
 *
 * description: This program is to demonstrate the search on string-like data by the GPU. More description of the parameter configuration please refer to example.cu file
 */

#include "GPUGenie.h" //for ide: change from "GPUGenie.h" to "../src/GPUGenie.h"
#include <stdio.h>
#include <stdlib.h>
#include <vector>
#include <fstream>
#include <string>

using namespace GPUGenie;
using namespace std;


int main(int argc, char * argv[])//for ide: from main to main4
{
	Logger::set_level(Logger::DEBUG);
	std::vector<std::vector<int> > queries;
	std::vector<attr_t> multirange_queries;
	//std::vector<std::vector<int> > data;
	inv_table table;
    GPUGenie::GPUGenie_Config config;

	string  dataFile = "tweets_4k.dat";//for ide: from "sift_1k.csv" to "example/sift_1k.csv"
    string  queryFile= "tweets_4k.csv";

	//Data dimension
	//For string search, we use one-dimension-mulitple-values method,
	// i.e. there is only one dimension, all words are considered as the values. 
	//given a query, there can be multiple values for this dimension. 
	//This is like a bag-of-word model for string search
	config.dim = 14;

	//Points with dim counts lower than threshold will be discarded and not shown in topk.
	//It is implemented as a bitmap filter.
	//Set to 0 to disable the feature.
	//set to <0, to use adaptiveThreshold, the absolute value of count_threshold is the maximum possible count sotred in the bitmap
	config.count_threshold = -14;

	//Number of topk items desired for each query.
	config.num_of_topk = 10;

	//if config.hashtable_size<=2, the hashtable_size means ratio against data size
			//Hash Table size is set as: config.hashtable_size (i.e.) ratio X data size.
			//Topk items will be generated from the hash table so it must be sufficiently large.
			//If set too small, the program will attempt to increase the size by 0.1f as many times
			//as possible. So to reduce the attempt time waste, please set to 1.0f if memory allows.
	//if config.hashtable_size>2, the hashtable_size means the size of the hashtable,
			//this is useful when using adaptiveThreshold (i.e. config.count_threshold <0), where the
			//hash_table size is usually set as: maximum_countXconfig.num_of_topkx1.5 (where 1.5 is load factor for hashtable).
	config.hashtable_size = 14*config.num_of_topk*1.5;//960

	//Query radius from the data point bucket expanding to upward and downward.
		//For tweets data, set it as 0, which means exact match
	//Will be overwritten by selectivity if use_adaptive_range is set.
	config.query_radius = 0;

	//Index of the GPU device to be used. If you only have one card, then set to 0.
	config.use_device = 0;

	//use_adaptive_range is not suitable for string search
	config.use_adaptive_range = false;
	config.selectivity = 0.0f;

	config.query_points = &queries;

	//if use_load_balance=false, config.multiplier and config.posting_list_max_length are not useful
	config.use_load_balance = true;
	//maximum number per posting list, if a keyword has a long posting list, we break it into sublists, and this parameter defines the maximum length of sub-list
	config.posting_list_max_length = 64000;
	config.multiplier = 1.5f;//config.multiplier*config.posting_list_max_length is  maximum number of elements processed by one block

	config.use_multirange = false;

    config.data_type = 1;
    config.search_type = 1;
    config.max_data_size = 0;

    config.num_of_queries = 10;

    read_file(dataFile.c_str(), &config.data, config.item_num, &config.index, config.row_num);
	if(config.use_multirange)
	{
		read_query(multirange_queries, queryFile.c_str(), -1);
		config.multirange_query_points = &multirange_queries;
	} else {
		read_file(queries, queryFile.c_str(), config.num_of_queries);
		config.query_points = &queries;
	}

	/**
	* @brief Search on the inverted index and save the result in result
	* bijectMap means building each ordered pair/keyword is also transformed by a bijection map. (Different from the default method, where the
	* keyword is a combination of dimension and value
	* Previous name: knn_search_tweets()
	*
	*/

    //example of writing and reading operations
    inv_table * __table = NULL;
    preprocess_for_knn_binary(config, __table);

    u64 s1 = getTime();
    inv_table::write("table.dat", __table);
    u64 e1 = getTime();

    double time1 = getInterval(s1, e1);
    cout<<"time1 = "<<time1<<endl;
    delete[] __table;
  
  //  unsigned int table_num = 1;
    inv_table * _table = NULL;
    u64 s2 = getTime();
    inv_table::read("table.dat", _table);
    u64 e2 = getTime();

    double time2 = getInterval(s2, e2);
    cout<<"time2 = "<<time2<<endl;
	std::vector<int> result, result_count;

	Logger::log(Logger::INFO, " example_sift Launching knn functions...");

	u64 start = getTime();
	//GPUGenie::knn_search(result, result_count, config);
    knn_search_after_preprocess(config, _table, result, result_count);
	u64 end = getTime();
	double elapsed = getInterval(start, end);

	Logger::log(Logger::VERBOSE, ">>>>>>> [time profiling]: Total Time Elapsed: %fms. <<<<<<<", elapsed);

    GPUGenie::reset_device();
	for(int i = 0; i < 5; ++i)

	{
		printf("Query %d result is: \n\t", i);
		for (int j = 0; j < 10; ++j)
		{
			printf("%d:%d, ", result[i * config.num_of_topk + j], result_count[i * config.num_of_topk + j]);
		}
		printf("\n");
	}

	Logger::exit();
    
    free(config.data);
    free(config.index);

    return 0;
}


