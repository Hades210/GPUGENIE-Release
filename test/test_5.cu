/** Name: test_5.cu
 * Description:
 * focus on save_to_gpu function
 *   sift data
 *   data is from csv file
 *   query is from csv file, single range
 *
 *
 */


#include "GPUGenie.h"

#include <assert.h>
#include <vector>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
using namespace std;
using namespace GPUGenie;

int main(int argc, char* argv[])
{
    string dataFile = "../static/sift_20.csv";
    string queryFile = "../static/sift_20.csv";
    vector<vector<int> > queries;
    vector<vector<int> > data;
    inv_table * table = NULL;
    GPUGenie_Config config;

    config.dim = 5;
    config.count_threshold = 14;
    config.num_of_topk = 5;
    config.hashtable_size = 14*config.num_of_topk*1.5;
    config.query_radius = 0;
    config.use_device = 0;
    config.use_adaptive_range = false;
    config.selectivity = 0.0f;

    config.query_points = &queries;
    config.data_points = &data;

    config.use_load_balance = false;
    config.posting_list_max_length = 6400;
    config.multiplier = 1.5f;
    config.use_multirange = false;

    config.data_type = 0;
    config.search_type = 0;
    config.max_data_size = 0;
    config.num_of_queries = 3;

    config.save_to_gpu = true;

    read_file(data, dataFile.c_str(), -1);
    read_file(queries, queryFile.c_str(), config.num_of_queries);

    preprocess_for_knn_csv(config, table);

    /**test for table*/
    vector<int>& inv = *table[0].inv();
    assert(inv[0] == 8);
    assert(inv[1] == 9);
    assert(inv[2] == 7);
    assert(inv[3] == 0);
    assert(inv[4] == 2);
    assert(inv[5] == 4);

    vector<int> result;
    vector<int> result_count;
    knn_search_after_preprocess(config, table, result, result_count);

    assert(result[0] == 0);
    assert(result_count[0] == 5);

    assert(result[1] == 4);
    assert(result_count[1] == 2);

    assert(result[5] == 1);
    assert(result_count[5] == 5);
    
    assert(result[10] == 2);
    assert(result_count[10] == 5);

    int i_size = inv.size();
    int* _inv = (int*)malloc(sizeof(int)*i_size);
    cudaCheckErrors(hipMemcpy(_inv, table[0].d_inv_p, sizeof(int)*i_size, hipMemcpyDeviceToHost));
    table[0].clear_gpu_mem();

    assert(_inv[0] == 8);
    assert(_inv[1] == 9);
    assert(_inv[2] == 7);
    assert(_inv[3] == 0);
    assert(_inv[4] == 2);
    assert(_inv[5] == 4);

    reset_device();
    free(_inv);

    delete[] table;
    return 0;
}
