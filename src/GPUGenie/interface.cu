#include "hip/hip_runtime.h"
/*! \file interface.cu
 *  \brief Implementation for interface declared in interface.h
 */

#include "interface.h"

#include <stdio.h>
#include <stdlib.h>

#include <iostream>
#include <sstream>
#include <fstream>

#include <string>
#include <sys/time.h>
#include <ctime>
#include <map>
#include <vector>
#include <algorithm>
#include <string>

#include <thrust/system_error.h>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include "Logger.h"
#include "Timing.h"

using namespace GPUGenie;
using namespace std;


bool GPUGenie::preprocess_for_knn_csv(GPUGenie_Config& config,
		inv_table * &_table)
{

    if (config.data_points->size() > 0)
    {
        _table = new inv_table[1];
        _table[0].set_table_index(0);
        _table[0].set_total_num_of_table(1);
        Logger::log(Logger::DEBUG, "build from data_points...");
        switch (config.search_type)
        {
        case 0:
            load_table(_table[0], *(config.data_points), config);
            break;
        case 1:
            load_table_bijectMap(_table[0], *(config.data_points), config);
            break;
        default:
            throw GPUGenie::cpu_runtime_error("Unrecognised search type!");
        }
    }
    else
    {
        throw GPUGenie::cpu_runtime_error("no data input!");
    }
	return true;
}

bool GPUGenie::preprocess_for_knn_binary(GPUGenie_Config& config, inv_table * &_table)
{
    if (config.item_num != 0 && config.index != NULL && config.item_num != 0 && config.row_num != 0)
    {
        _table = new inv_table[1];
        _table[0].set_table_index(0);
        _table[0].set_total_num_of_table(1);
        Logger::log(Logger::DEBUG, "build from data array...");
        switch (config.search_type)
        {
        case 0:
            load_table(_table[0], config.data, config.item_num, config.index,
                    config.row_num, config);
            break;
        case 1:
            load_table_bijectMap(_table[0], config.data, config.item_num,
                    config.index, config.row_num, config);
            break;
        }
    }
    else
    {
        throw GPUGenie::cpu_runtime_error("no data input!");
    }
	return true;
}

void GPUGenie::knn_search_after_preprocess(GPUGenie_Config& config,
		inv_table * &_table, std::vector<int>& result,
		std::vector<int>& result_count)
{
    vector<query> queries;
    queries.clear();
    load_query(_table[0], queries, config);

    knn_search(_table[0], queries, result, result_count,config);

}
void GPUGenie::load_table(inv_table& table,
		std::vector<std::vector<int> >& data_points, GPUGenie_Config& config)
{
	inv_list list;
	u32 i, j;

	Logger::log(Logger::DEBUG, "Data row size: %d. Data Row Number: %d.",
			data_points[0].size(), data_points.size());
	u64 starttime = getTime();

	for (i = 0; i < data_points[0].size(); ++i)
	{
		std::vector<int> col;
		col.reserve(data_points.size());
		for (j = 0; j < data_points.size(); ++j)
		{
			col.push_back(data_points[j][i]);
		}
		list.invert(col);
		table.append(list);
	}

	table.build(config.posting_list_max_length, config.use_load_balance);

	u64 endtime = getTime();
	double timeInterval = getInterval(starttime, endtime);
	Logger::log(Logger::DEBUG,
			"Before finishing loading. i_size():%d, m_size():%d.",
			table.i_size(), table.m_size());
	Logger::log(Logger::VERBOSE,
			">>>>[time profiling]: loading index takes %f ms<<<<",
			timeInterval);

}

void GPUGenie::load_table(inv_table& table, int *data, unsigned int item_num,
		unsigned int *index, unsigned int row_num, GPUGenie_Config& config)
{
	inv_list list;
	u32 i, j;

	unsigned int row_size;
	unsigned int index_start_pos = 0;
	if (row_num == 1)
		row_size = item_num;
	else
		row_size = index[1] - index[0];

	index_start_pos = index[0];

	Logger::log(Logger::DEBUG, "Data row size: %d. Data Row Number: %d.",
			index[1], row_num);
	u64 starttime = getTime();

	for (i = 0; i < row_size; ++i)
	{
		std::vector<int> col;
		col.reserve(row_num);
		for (j = 0; j < row_num; ++j)
		{
			col.push_back(data[index[j] + i - index_start_pos]);
		}
		list.invert(col);
		table.append(list);
	}

	table.build(config.posting_list_max_length, config.use_load_balance);

	u64 endtime = getTime();
	double timeInterval = getInterval(starttime, endtime);
	Logger::log(Logger::DEBUG,
			"Before finishing loading. i_size() : %d, m_size() : %d.",
			table.i_size(), table.m_size());
	Logger::log(Logger::VERBOSE,
			">>>>[time profiling]: loading index takes %f ms<<<<",
			timeInterval);

}

void GPUGenie::load_query(inv_table& table, std::vector<query>& queries,
		GPUGenie_Config& config)
{
	if (config.use_multirange)
	{
		load_query_multirange(table, queries, config);
	}
	else
	{
		load_query_singlerange(table, queries, config);
	}
}

//Read new format query data
//Sample data format
//qid dim value selectivity weight
// 0   0   15     0.04        1
// 0   1   6      0.04        1
// ....
void GPUGenie::load_query_multirange(inv_table& table,
		std::vector<query>& queries, GPUGenie_Config& config)
{
	queries.clear();
	map<int, query> query_map;
	int qid, dim, val;
	float sel, weight;
	for (unsigned int iq = 0; iq < config.multirange_query_points->size(); ++iq)
	{
		attr_t& attr = (*config.multirange_query_points)[iq];

		qid = attr.qid;
		dim = attr.dim;
		val = attr.value;
		weight = attr.weight;
		sel = attr.sel;
		if (query_map.find(qid) == query_map.end())
		{
			query q(table, qid);
			q.topk(config.num_of_topk);
			if (config.selectivity > 0.0f)
			{
				q.selectivity(config.selectivity);
			}
			if (config.use_load_balance)
			{
				q.use_load_balance = true;
			}
			query_map[qid] = q;

		}
		query_map[qid].attr(dim, val, weight, sel);
	}
	for (std::map<int, query>::iterator it = query_map.begin();
			it != query_map.end() && queries.size() < (unsigned int) config.num_of_queries;
			++it)
	{
		query& q = it->second;
		q.apply_adaptive_query_range();
		queries.push_back(q);
	}

	Logger::log(Logger::INFO, "Finish loading queries!");
	Logger::log(Logger::DEBUG, "%d queries are loaded.", queries.size());

}
void GPUGenie::load_query_singlerange(inv_table& table,
		std::vector<query>& queries, GPUGenie_Config& config)
{

	Logger::log(Logger::DEBUG, "Table dim: %d.", table.m_size());
	u64 starttime = getTime();

	u32 i, j;
	int value;
	int radius = config.query_radius;
	std::vector<std::vector<int> >& query_points = *config.query_points;
	for (i = 0; i < query_points.size() && i < (unsigned int) config.num_of_queries; ++i)
	{
		query q(table, i);

		for (j = 0;
				j < query_points[i].size()
						&& (config.search_type == 1 || j < (unsigned int) config.dim); ++j)
		{
			value = query_points[i][j];
			if (value < 0)
			{
				continue;
			}

			q.attr(config.search_type == 1 ? 0 : j,
					value - radius < 0 ? 0 : value - radius, value + radius,
					GPUGENIE_DEFAULT_WEIGHT);
		}

		q.topk(config.num_of_topk);
		q.selectivity(config.selectivity);
		if (config.use_adaptive_range)
		{
			q.apply_adaptive_query_range();
		}
		if (config.use_load_balance)
		{
			q.use_load_balance = true;
		}

		queries.push_back(q);
	}

	u64 endtime = getTime();
	double timeInterval = getInterval(starttime, endtime);
	Logger::log(Logger::INFO, "%d queries are created!", queries.size());
	Logger::log(Logger::VERBOSE,
			">>>>[time profiling]: loading query takes %f ms<<<<",
			timeInterval);
}

void GPUGenie::load_table_bijectMap(inv_table& table,
		std::vector<std::vector<int> >& data_points, GPUGenie_Config& config)
{
	u64 starttime = getTime();

	inv_list list;
	list.invert_bijectMap(data_points);
	table.append(list);
	table.build(config.posting_list_max_length, config.use_load_balance);


	u64 endtime = getTime();
	double timeInterval = getInterval(starttime, endtime);
	Logger::log(Logger::DEBUG,
			"Before finishing loading. i_size():%d, m_size():%d.",
			table.i_size(), table.m_size());
	Logger::log(Logger::VERBOSE,
			">>>>[time profiling]: loading index takes %f ms (for one dim multi-values)<<<<",
			timeInterval);

}

void GPUGenie::load_table_bijectMap(inv_table& table, int *data,
		unsigned int item_num, unsigned int *index, unsigned int row_num,
		GPUGenie_Config& config)
{

	u64 starttime = getTime();

	inv_list list;
	list.invert_bijectMap(data, item_num, index, row_num);

	table.append(list);
	table.build(config.posting_list_max_length, config.use_load_balance);


	u64 endtime = getTime();
	double timeInterval = getInterval(starttime, endtime);
	Logger::log(Logger::DEBUG,
			"Before finishing loading. i_size():%d, m_size():%d.",
			table.i_size(), table.m_size());
	Logger::log(Logger::VERBOSE,
			">>>>[time profiling]: loading index takes %f ms (for one dim multi-values)<<<<",
			timeInterval);

}

void GPUGenie::knn_search_for_binary_data(std::vector<int>& result,
		std::vector<int>& result_count, GPUGenie_Config& config)
{
	
	inv_table *_table = NULL;

	preprocess_for_knn_binary(config, _table);

	knn_search_after_preprocess(config, _table, result, result_count);

	delete[] _table;
}

void GPUGenie::knn_search_for_csv_data(std::vector<int>& result,
		std::vector<int>& result_count, GPUGenie_Config& config)
{
	inv_table *_table = NULL;

	Logger::log(Logger::VERBOSE, "Starting preprocessing!");
	preprocess_for_knn_csv(config, _table);

	Logger::log(Logger::VERBOSE, "preprocessing finished!");

	knn_search_after_preprocess(config, _table, result, result_count);

	delete[] _table;
}

void GPUGenie::knn_search(std::vector<int>& result, GPUGenie_Config& config)
{
	std::vector<int> result_count;
	knn_search(result, result_count, config);
}

void GPUGenie::knn_search(std::vector<int>& result,
		std::vector<int>& result_count, GPUGenie_Config& config)
{
	try{
		u64 starttime = getTime();
		switch (config.data_type)
		{
		case 0:
			Logger::log(Logger::INFO, "search for csv data!");
			knn_search_for_csv_data(result, result_count, config);
			cout<<"knn for csv finished!"<<endl;
            break;
		case 1:
			Logger::log(Logger::INFO, "search for binary data!");
			knn_search_for_binary_data(result, result_count, config);
			break;
		default:
			throw GPUGenie::cpu_runtime_error("Please check data type in config\n");
		}

		u64 endtime = getTime();
		double elapsed = getInterval(starttime, endtime);

		Logger::log(Logger::VERBOSE,
				">>>>[time profiling]: knn_search totally takes %f ms (building query+match+selection)<<<<",
				elapsed);
	}
	catch (thrust::system::system_error &e){
        cout<<"system_error : "<<e.what()<<endl;
		throw GPUGenie::gpu_runtime_error(e.what());
	} catch (GPUGenie::gpu_bad_alloc &e){
        cout<<"bad_alloc"<<endl;
		throw e;
	} catch (GPUGenie::gpu_runtime_error &e){
		cout<<"run time error"<<endl;
        throw e;
	} catch(std::bad_alloc &e){
        cout<<"cpu bad alloc"<<endl;
		throw GPUGenie::cpu_bad_alloc(e.what());
	} catch(std::exception &e){
        cout<<"cpu runtime"<<endl;
		throw GPUGenie::cpu_runtime_error(e.what());
	} catch(...){
        cout<<"other error"<<endl;
		std::string msg = "Warning: Unknown Exception! Please try again or contact the author.\n";
		throw GPUGenie::cpu_runtime_error(msg.c_str());
	}
}

void GPUGenie::knn_search(inv_table& table, std::vector<query>& queries,
		std::vector<int>& h_topk, std::vector<int>& h_topk_count,
		GPUGenie_Config& config)
{
	int device_count, hashtable_size;
	cudaCheckErrors(hipGetDeviceCount(&device_count));
	if (device_count == 0)
	{
		throw GPUGenie::cpu_runtime_error("NVIDIA CUDA-SUPPORTED GPU NOT FOUND! Program aborted..");
	}
	else if (device_count <= config.use_device)
	{
		Logger::log(Logger::INFO,
				"[Info] Device %d not found! Changing to %d...",
				config.use_device, GPUGENIE_DEFAULT_DEVICE);
		config.use_device = GPUGENIE_DEFAULT_DEVICE;
	}
	cudaCheckErrors(hipSetDevice(config.use_device));

	Logger::log(Logger::INFO, "Using device %d...", config.use_device);
	Logger::log(Logger::DEBUG, "table.i_size():%d, config.hashtable_size:%f.",
			table.i_size(), config.hashtable_size);

	if (config.hashtable_size <= 2)
	{
		hashtable_size = table.i_size() * config.hashtable_size + 1;
	}
	else
	{
		hashtable_size = config.hashtable_size;
	}
	thrust::device_vector<int> d_topk, d_topk_count;

	int max_load = config.multiplier * config.posting_list_max_length + 1;

	Logger::log(Logger::DEBUG, "max_load is %d", max_load);

	GPUGenie::knn_bijectMap(
			table, //basic API, since encode dimension and value is also finally transformed as a bijection map
			queries, d_topk, d_topk_count, hashtable_size, max_load,
			config.count_threshold);

	Logger::log(Logger::INFO, "knn search is done!");
	Logger::log(Logger::DEBUG, "Topk obtained: %d in total.", d_topk.size());

	h_topk.resize(d_topk.size());
	h_topk_count.resize(d_topk_count.size());

	thrust::copy(d_topk.begin(), d_topk.end(), h_topk.begin());
	thrust::copy(d_topk_count.begin(), d_topk_count.end(),
			h_topk_count.begin());
}


void GPUGenie::reset_device()
{
    hipDeviceReset();
}
