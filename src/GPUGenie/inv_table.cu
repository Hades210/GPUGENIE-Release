#include "hip/hip_runtime.h"
/*! \file inv_table.cu
 *  \brief Implementation of class inv_table 
 *  declared in file inv_table.h
 */


#include <stdio.h>
#include <fstream>
#include <exception>
#include <iostream>
#include "Logger.h"
#include "genie_errors.h"

#include "inv_table.h"
using namespace std;
using namespace GPUGenie;


bool GPUGenie::inv_table::cpy_data_to_gpu()
{
	try{
		cudaCheckErrors(hipMalloc(&d_inv_p, sizeof(int) * _inv.size()));
		cudaCheckErrors(hipMemcpy(d_inv_p, &_inv[0], sizeof(int) * _inv.size(),hipMemcpyHostToDevice));
	} catch(std::bad_alloc &e){
		throw(GPUGenie::gpu_bad_alloc(e.what()));
	}

	return true;
}

void GPUGenie::inv_table::clear()
{
	_build_status = not_builded;
	_inv_lists.clear();
	_ck.clear();
	_inv.clear();
	clear_gpu_mem();
}

GPUGenie::inv_table::~inv_table()
{
}

void GPUGenie::inv_table::clear_gpu_mem()
{
	hipFree(d_inv_p);
}

bool GPUGenie::inv_table::empty()
{
	return _size == -1;
}

int GPUGenie::inv_table::m_size()
{
    return _dim_size;
	//return _inv_lists.size();
}

int GPUGenie::inv_table::i_size()
{
	return _size <= -1 ? 0 : _size;
}

int GPUGenie::inv_table::shifter()
{
	return _shifter;
}

void GPUGenie::inv_table::append(inv_list& inv)
{
	if (_size == -1 || _size == inv.size())
	{
		_build_status = not_builded;
		_size = inv.size();
		_inv_lists.push_back(inv);
        
        _dim_size = _inv_lists.size();
        
        //get size for every posting list

        vector<int> line;
        for(int i = 0 ; i < inv.value_range() ; ++i)
            line.push_back(inv.index(i+inv.min())->size());
        posting_list_size.push_back(line);

        //get upperbound and lowerbound for every inv_list
        inv_list_upperbound.push_back(inv.max());
        inv_list_lowerbound.push_back(inv.min());
	}
}

void GPUGenie::inv_table::append(inv_list* inv)
{
	if (inv != NULL)
	{
		append(*inv);
	}
}

int
GPUGenie::inv_table::get_posting_list_size(int attr_index, int value)
{
    if((unsigned int)attr_index<posting_list_size.size() && value>=inv_list_lowerbound[attr_index] && value<=inv_list_upperbound[attr_index])
        return posting_list_size[attr_index][value-inv_list_lowerbound[attr_index]];
    else
        return 0;
}

bool
GPUGenie::inv_table::list_contain(int attr_index, int value)
{
    if(value <= inv_list_upperbound[attr_index] && value >= inv_list_lowerbound[attr_index])
        return true;
    else
        return false;
}



int
GPUGenie::inv_table::get_upperbound_of_list(int index)
{
    if((unsigned int)index < inv_list_upperbound.size())
        return inv_list_upperbound[index];
    else
        return -1;
}

int
GPUGenie::inv_table::get_lowerbound_of_list(int index)
{
    if((unsigned int)index < inv_list_lowerbound.size())
        return inv_list_lowerbound[index];
    else
        return -1;
}


void
GPUGenie::inv_table::set_table_index(int index)
{
    table_index = index;
}
void
GPUGenie::inv_table::set_total_num_of_table(int num)
{
    total_num_of_table = num;
}

int
GPUGenie::inv_table::get_table_index()
{
    return table_index;
}
    
int
GPUGenie::inv_table::get_total_num_of_table()
{
    return total_num_of_table;
}

GPUGenie::inv_table::status GPUGenie::inv_table::build_status()
{
	return _build_status;
}

vector<inv_list>*
GPUGenie::inv_table::inv_lists()
{
	return &_inv_lists;
}

vector<int>*
GPUGenie::inv_table::ck()
{
	return &_ck;
}

vector<int>*
GPUGenie::inv_table::inv()
{
	return &_inv;
}

vector<int>*
GPUGenie::inv_table::inv_index()
{
	return &_inv_index;
}

vector<int>*
GPUGenie::inv_table::inv_pos()
{
	return &_inv_pos;
}


void
GPUGenie::inv_table::build(u64 max_length, bool use_load_balance)
{
	_ck.clear(), _inv.clear();
	_inv_index.clear();
	_inv_pos.clear();
    if(!use_load_balance)
    {
        max_length = (u64)0 - (u64)1;
    }
    unsigned int last;
	int key, dim, value;
	for (unsigned int i = 0; i < _inv_lists.size(); i++)
	{
		dim = i << _shifter;
		for (value = _inv_lists[i].min(); value <= _inv_lists[i].max(); value++)
		{
			key = dim + value - _inv_lists[i].min();
			vector<int>& index = *_inv_lists[i].index(value);

			if (_ck.size() <= (unsigned int) key)
			{
				last = _ck.size();
				_ck.resize(key + 1);
				_inv_index.resize(key + 1);
				for (; last < _ck.size(); last++)
				{
					_ck[last] = _inv.size();
					_inv_index[last] = _inv_pos.size();
				}
			}
			for (unsigned int j = 0; j < index.size(); j++)
			{
                if (j % max_length == 0)
				{
					_inv_pos.push_back(_inv.size());
				}
				_inv.push_back(index[j]);
				_ck[key] = _inv.size();
			}

		}

	}
	_inv_index.push_back(_inv_pos.size());
	_inv_pos.push_back(_inv.size());

	_build_status = builded;
	Logger::log(Logger::DEBUG, "inv_index size %d:", _inv_index.size());
	Logger::log(Logger::DEBUG, "inv_pos size %d:", _inv_pos.size());
	Logger::log(Logger::DEBUG, "inv size %d:", _inv.size());
}



bool
GPUGenie::inv_table::write_to_file(ofstream& ofs)
{
    if(_build_status == not_builded)
        return false;

    ofs.write((char*)&table_index, sizeof(int));
    ofs.write((char*)&total_num_of_table, sizeof(int));
    ofs.write((char*)&_shifter, sizeof(int));
    ofs.write((char*)&_size, sizeof(int));
    ofs.write((char*)&_dim_size, sizeof(int));
    int temp_status = _build_status;
    ofs.write((char*)&temp_status, sizeof(int));

    unsigned int _ck_size = _ck.size();
    unsigned int _inv_size = _inv.size();
    unsigned int _inv_index_size = _inv_index.size();
    unsigned int _inv_pos_size = _inv_pos.size();
    
    ofs.write((char*)&_ck_size, sizeof(unsigned int));
    ofs.write((char*)&_inv_size, sizeof(unsigned int));
    ofs.write((char*)&_inv_index_size, sizeof(unsigned int));
    ofs.write((char*)&_inv_pos_size, sizeof(unsigned int));

    ofs.write((char*)&_ck[0], _ck_size*sizeof(int));
    ofs.write((char*)&_inv[0], _inv_size*sizeof(int));
    ofs.write((char*)&_inv_index[0],_inv_index_size*sizeof(int));
    ofs.write((char*)&_inv_pos[0], _inv_pos_size*sizeof(int));

    unsigned int _list_upperbound_size = inv_list_upperbound.size();
    unsigned int _list_lowerbound_size = inv_list_lowerbound.size();

    ofs.write((char*)&_list_upperbound_size, sizeof(unsigned int));
    ofs.write((char*)&_list_lowerbound_size, sizeof(unsigned int));

    ofs.write((char*)&inv_list_upperbound[0], _list_upperbound_size*sizeof(int));
    ofs.write((char*)&inv_list_lowerbound[0], _list_lowerbound_size*sizeof(int));

    //write posting list size
    unsigned int num_of_attr = posting_list_size.size();
    ofs.write((char*)&num_of_attr, sizeof(unsigned int));
    for(unsigned int i=0 ; i<num_of_attr ; ++i)
    {
         unsigned int value_range_size = posting_list_size[i].size();
         ofs.write((char*)&value_range_size, sizeof(unsigned int));
         ofs.write((char*)&posting_list_size[i][0], value_range_size*sizeof(int));
    }
    

    if(table_index == total_num_of_table - 1)
        ofs.close();
    return true;
}


bool
GPUGenie::inv_table::read_from_file(ifstream& ifs)
{
    
    ifs.read((char*)&table_index, sizeof(int));
    ifs.read((char*)&total_num_of_table, sizeof(int));
    ifs.read((char*)&_shifter, sizeof(int));
    ifs.read((char*)&_size, sizeof(int));
    ifs.read((char*)&_dim_size, sizeof(int));
    int temp_status;
    ifs.read((char*)&temp_status, sizeof(int));
    _build_status = static_cast<status>(temp_status);


    unsigned int _ck_size;
    unsigned int _inv_size;
    unsigned int _inv_index_size;
    unsigned int _inv_pos_size;

    ifs.read((char*)&_ck_size, sizeof(unsigned int));
    ifs.read((char*)&_inv_size, sizeof(unsigned int));
    ifs.read((char*)&_inv_index_size, sizeof(unsigned int));
    ifs.read((char*)&_inv_pos_size, sizeof(unsigned int));

    _ck.resize(_ck_size);
    _inv.resize(_inv_size);
    _inv_index.resize(_inv_index_size);
    _inv_pos.resize(_inv_pos_size);

    ifs.read((char*)&_ck[0], _ck_size*sizeof(int));
    ifs.read((char*)&_inv[0], _inv_size*sizeof(int));
    ifs.read((char*)&_inv_index[0],_inv_index_size*sizeof(int));
    ifs.read((char*)&_inv_pos[0], _inv_pos_size*sizeof(int));
    
    unsigned int _list_upperbound_size;
    unsigned int _list_lowerbound_size;

    ifs.read((char*)&_list_upperbound_size, sizeof(unsigned int));
    ifs.read((char*)&_list_lowerbound_size, sizeof(unsigned int));

    inv_list_upperbound.resize(_list_upperbound_size);
    inv_list_lowerbound.resize(_list_lowerbound_size);
    ifs.read((char*)&inv_list_upperbound[0], _list_upperbound_size*sizeof(int));
    ifs.read((char*)&inv_list_lowerbound[0], _list_lowerbound_size*sizeof(int));

    unsigned int num_of_attr;
    ifs.read((char*)&num_of_attr, sizeof(unsigned int));
    posting_list_size.resize(num_of_attr);
    for(unsigned int i=0 ; i<num_of_attr ; ++i)
    {
         unsigned int value_range_size;
         ifs.read((char*)&value_range_size, sizeof(unsigned int));
         posting_list_size[i].resize(value_range_size);
         ifs.read((char*)&posting_list_size[i][0], value_range_size*sizeof(int));
    }

    if(table_index == total_num_of_table-1)
        ifs.close();
    
    return true;
}


bool
GPUGenie::inv_table::write(const char* filename, inv_table*& table)
{
    
    int _table_index = table[0].get_table_index();
    if(_table_index != 0)
        return false;
    
    ofstream _ofs(filename, ios::binary|ios::trunc|ios::out);
    if(!_ofs.is_open())
        return false;
    int _total_num_of_table = table[0].get_total_num_of_table();
    bool success;
    for(int i=0; i<_total_num_of_table; ++i)
    {
        success = table[i].write_to_file(_ofs);
    }
    

    return !_ofs.is_open() && success;

    
}

bool
GPUGenie::inv_table::read(const char* filename, inv_table*& table)
{
    ifstream ifs(filename, ios::binary|ios::in);
    if(!ifs.is_open())
        return false;
    
    int _table_index, _total_num_of_table;
    ifs.read((char*)&_table_index, sizeof(int));
    ifs.read((char*)&_total_num_of_table, sizeof(int));
    ifs.close();
    if(_table_index!=0 || _total_num_of_table<1)
        return false;
    
    table = new inv_table[_total_num_of_table];
    ifstream _ifs(filename, ios::binary|ios::in);
    
    bool success;
    for(int i=0 ; i<_total_num_of_table ; ++i)
    {
         success = table[i].read_from_file(_ifs);
    }
    return !_ifs.is_open() && success;
}
